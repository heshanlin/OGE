#include "hip/hip_runtime.h"
#include "global.h"
__device__ float wd(char x, char y, float match, float mismatch)
{
    return (x == y ? match : mismatch);
}


__device__ SEQ_ALIGN makeElement_d(int state, int segment_length)
{
    SEQ_ALIGN t;
    t.o = '\0';
    switch(state) {
    case MSTATE:
        t.o = 'M';
        break;
    case ISTATE:
        t.o = 'I';
        break;
    case DSTATE:
        t.o = 'D';
        break;
    case 'S':
        t.o = 'S';
        break;
    }
    t.length = segment_length;
    return t;
}

__device__ void compare_t(SEQ_ALIGN *t, int &p_flag, int &c_flag, int &seg_length, int &n_Cigar)
{
    if(p_flag != c_flag) {
        *t = makeElement_d(p_flag, seg_length);
        seg_length = 1;
        p_flag = c_flag;
        n_Cigar++;
    } else {
        seg_length++;
    }
}

__global__ void MatchString(
    char *pathFlags,
    char *extFlags,
    char  *instr1Ds,
    char  *instr2Ds,
    int *a_offset,
    int *b_offset,
    float open_penalty,
    float ext_penalty,
    float match,
    float mismatch,
    int *rowNums,
    int *columnNums,
    SEQ_ALIGN *backtrace,
    int *n_C,
    int *alignment_offset,
    int maxMatrix)

{
    int npos, npos_b;
    int npreposngap, npreposhgap, npreposvgap;
    int npreposngap_b, npreposhgap_b, npreposvgap_b;
    int indexi, indexj;
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    int preOffsetArray[3] = {2, -1, -1};
    int prePreOffsetArray[3] = {1, 1, -2};

    float fdistngap, fdisthgap, fdistvgap;
    float ext_dist;
    float fmaxdist;

    __shared__ float str_fngapdistp_s[128 * 3];
    __shared__ float str_fhgapdistp_s[128 * 3];
    __shared__ float str_fvgapdistp_s[128 * 3];

    __shared__ float maxscore;
    __shared__ int nposi_s;
    __shared__ int nposj_s;

    for(int tt = 0; tt < 3; tt++) {
        int offset = tt * 128;
        str_fngapdistp_s[tid + offset] = 0.0f;
        str_fhgapdistp_s[tid + offset] = 0.0f;
        str_fvgapdistp_s[tid + offset] = 0.0f;

    }

    SEQ_ALIGN *t = backtrace + bid * 100;

    char  *instr1D = instr1Ds + a_offset[bid];
    char  *instr2D = instr2Ds + b_offset[bid];
    char  *pathFlag = pathFlags + maxMatrix * bid;
    char  *extFlag = extFlags + maxMatrix * bid;
    int seq1Pos = 0;
    int seq2Pos = 1;
    int nOffset = 1;
    int threadsOfLaunch = 0;
    int rowNum = rowNums[bid] + 1;
    int columnNum = columnNums[bid] + 1;
    int launchNum = rowNum + columnNum - 1;

    if(tid == 0) {
        maxscore = 0;
    }

    for(int launchNo = 2; launchNo < launchNum; launchNo++) {
        if(launchNo <= rowNum) {
            seq1Pos++;
            if(launchNo <= columnNum) {
                threadsOfLaunch++;
            }
        } else {
            seq2Pos++;
            nOffset++;
            threadsOfLaunch--;
        }


        int nposi = seq1Pos;
        int nposj = seq2Pos;
        int   launchNoMod3 = launchNo % 3;


        if(tid < threadsOfLaunch) {
            npos = launchNoMod3 * columnNum + nOffset + tid;
            indexj = nposj + tid;
            indexi = nposi - tid;
            npreposhgap = npos + preOffsetArray[launchNoMod3] * columnNum;
            npreposvgap = npreposhgap - 1;
            npreposngap = npos + prePreOffsetArray[launchNoMod3] * columnNum - 1;


            npos_b = indexi * columnNum + indexj;
            npreposngap_b = (indexi - 1) * columnNum + indexj - 1;
            npreposhgap_b = npreposngap_b + 1;
            npreposvgap_b = npreposngap_b + columnNum;


            fdistngap = str_fngapdistp_s[npreposngap] + wd(instr1D[indexi - 1], instr2D[indexj - 1], match, mismatch);
            fdistvgap = str_fngapdistp_s[npreposvgap] + open_penalty;
            ext_dist  = str_fvgapdistp_s[npreposvgap] + ext_penalty;

            if(fdistvgap <= ext_dist && indexj > 1) {
                fdistvgap = ext_dist;
                pathFlag[npreposvgap_b] += 8;
            }

            fdisthgap = str_fngapdistp_s[npreposhgap] + open_penalty;
            ext_dist  = str_fhgapdistp_s[npreposhgap] + ext_penalty;

            if(fdisthgap <= ext_dist && indexi > 1) {
                fdisthgap = ext_dist;
                extFlag[npreposhgap_b] = 1;
            }

            str_fhgapdistp_s[npos] = fdisthgap;
            str_fvgapdistp_s[npos] = fdistvgap;

            int step_down = fdistvgap;
            int step_right = fdisthgap;
            int step_diag = fdistngap;

            if(step_down >= step_right) {
                if(step_down > step_diag) {
                    fmaxdist = step_down;
                    pathFlag[npos_b] = 3;
                } else {
                    fmaxdist = step_diag;
                    pathFlag[npos_b] = 2;
                }
            } else {
                if(step_right > step_diag) {
                    fmaxdist = step_right;
                    pathFlag[npos_b] = 1;
                } else {
                    fmaxdist = step_diag;
                    pathFlag[npos_b] = 2;
                }
            }


            str_fngapdistp_s[npos] = fmaxdist;
            if(fmaxdist >= maxscore && indexj == columnNum - 1) {
                maxscore = fmaxdist;
                nposj_s = columnNum - 1;
                nposi_s = indexi;
            }
            if((fmaxdist >= maxscore) && abs(rowNum - indexj) < abs(rowNum - nposj_s) && (nposi_s == rowNum - 1)) {
                maxscore = fmaxdist;
                nposj_s = indexj;
                nposi_s = rowNum - 1;
            }
        }
        __syncthreads();
    }


    if(tid == 0) {
        int i, j;
        int nlen;
        int npathflag;
        int n_Cigar = 0;
        i = nposi_s;
        j = nposj_s;
        npathflag = pathFlag[i * columnNum + j] & 0x3;
        nlen = 0;
        int prev_flag = npathflag;
        int segment_length = 0;
        while(1) {
            if(npathflag == 3) {
                nlen++;
                j--;
                compare_t(&t[n_Cigar], prev_flag, npathflag, segment_length, n_Cigar);
            } else if(npathflag == 1) {
                nlen++;
                i--;
                compare_t(&t[n_Cigar], prev_flag, npathflag, segment_length, n_Cigar);
            } else if(npathflag == 2) {
                nlen++;
                i--;
                j--;
                compare_t(&t[n_Cigar], prev_flag, npathflag, segment_length, n_Cigar);
            } else {
                break;
            }

            npos = i * columnNum + j;
            int nExtFlag = pathFlag[npos] / 4;

            if(npathflag == 3 && (nExtFlag == 2 || nExtFlag == 3)) {
                npathflag = 3;
            } else if(npathflag == 1 && extFlag[npos] == 1) {
                npathflag = 1;
            } else {
                npathflag = pathFlag[npos] & 0x3;
            }

            if(i == 0 || j == 0) {
                break;
            }

            if(npathflag == PATH_END) {
                break;
            }
        }

        compare_t(&t[n_Cigar], prev_flag, npathflag, segment_length, n_Cigar);
        alignment_offset[bid] = i - j;
        n_C[bid] = n_Cigar;

    }
}


