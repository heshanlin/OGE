#include "global.h"
#include "kernels.cu"
#include "swat_cuda.h"

struct timeval start_t1[10], end_t1[10];
#define TIMER_START(t) gettimeofday(&(start_t1[t]), NULL)
#define TIMER_END(t) gettimeofday(&(end_t1[t]), NULL)
#define MICRO_SECONDS(t) ((end_t1[t].tv_sec - start_t1[t].tv_sec)*1e6 + (end_t1[t].tv_usec - start_t1[t].tv_usec))

double swat_kernel_t = 0;
CigarOp makeElement(int state, int segment_length)
{
    char o = '\0';
    switch(state) {
    case MSTATE:
        o = 'M';
        break;
    case ISTATE:
        o = 'I';
        break;
    case DSTATE:
        o = 'D';
        break;
    case 'S':
        o = 'S';
        break;
    }
    return CigarOp(o, segment_length);
}


int swat(const byte a[], unsigned int an, const int a_offset[], const int ans[], const int n_a,
         const byte b[], unsigned int bn,  const int b_offset[], const int bns[], const int n_b,
         float openPenalty, float extensionPenalty, float match, float mismatch, vector< vector<CigarOp> >& Cigars, int *alignment_offset, int max_len)
{
    hipError_t cudaRes0, cudaRes1;
    int blockSize = 128;
    int subSequenceNum = n_b;
    char *pathFlagD;
    char *extFlagD;
    int maxElemNum = (max_len + 1);
    int maxMatrix = maxElemNum * 101;
    int *alignment_offsetD;

    for(int i = 0; i < n_a; i++) {
        if(ans[i] > MAX_LEN) {
            printf("an MAX: %d\n", ans[i]);
        }
    }

    for(int i = 0; i < n_b; i++) {
        if(bns[i] > MAX_LEN) {
            printf("bn MAX: %d\n", bns[i]);
        }
    }

    cudaRes0 = hipMalloc((void **)&alignment_offsetD, sizeof(int) * subSequenceNum);
    if(cudaRes0 != hipSuccess) {
        printf("Allocate sequence buffer on device error, %d\n",
               cudaRes0);
        return 1;
    }
    char *seq1D, *seq2D;

    cudaRes0 = hipMalloc((void **)&seq1D, sizeof(char) * an);
    cudaRes1 = hipMalloc((void **)&seq2D, sizeof(char) * bn);
    if(cudaRes0 != hipSuccess ||
            cudaRes1 != hipSuccess) {
        printf("Allocate sequence buffer on device error, %d, %d\n",
               cudaRes0,
               cudaRes1);
        return 1;
    }

    int *seq1_offD, *seq2_offD;

    cudaRes0 = hipMalloc((void **)&seq1_offD, sizeof(int) * subSequenceNum);
    cudaRes1 = hipMalloc((void **)&seq2_offD, sizeof(int) * subSequenceNum);
    if(cudaRes0 != hipSuccess ||
            cudaRes1 != hipSuccess) {
        printf("Allocate sequence buffer on device error, %d, %d\n",
               cudaRes0,
               cudaRes1);
        return 1;
    }


    int *rowNumsD, *columnNumsD;

    cudaRes0 = hipMalloc((void **)&rowNumsD, sizeof(int) * subSequenceNum);
    cudaRes1 = hipMalloc((void **)&columnNumsD, sizeof(int) * subSequenceNum);
    if(cudaRes0 != hipSuccess ||
            cudaRes1 != hipSuccess) {
        printf("Allocate sequence buffer on device error, %d, %d\n",
               cudaRes0,
               cudaRes1);
        return 1;
    }

    cudaRes0 = hipMalloc((void **)&pathFlagD, sizeof(char) * maxMatrix * subSequenceNum);
    cudaRes1 = hipMalloc((void **)&extFlagD,  sizeof(char) * maxMatrix * subSequenceNum);

    if((cudaRes0 != hipSuccess) ||
            (cudaRes1 != hipSuccess)) {
        printf("cuda allocate DP matrices on device  error: %d, %d\n",
               cudaRes0,
               cudaRes1);
        return 1;
    }

    SEQ_ALIGN *element, *elementD;
    element = new SEQ_ALIGN[100 * subSequenceNum];

    cudaRes0 = hipMalloc((void **)&elementD, sizeof(SEQ_ALIGN) * 100 * subSequenceNum);
    if(cudaRes0 != hipSuccess) {
        printf("Allocate maxInfo on device error!\n");
        return 1;
    }

    int *n_element, *n_elementD;
    n_element = new int[subSequenceNum];

    cudaRes0 = hipMalloc((void **)&n_elementD, sizeof(int) * subSequenceNum);
    if(cudaRes0 != hipSuccess) {
        printf("Allocate maxInfo on device error!\n");
        return 1;
    }

    //Initialize DP matrices
    hipMemset(pathFlagD, 0, maxMatrix * sizeof(char) * subSequenceNum);
    hipMemset(extFlagD,  0, maxMatrix * sizeof(char) * subSequenceNum);

    cudaRes0 = hipGetLastError();
    if(cudaRes0 != hipSuccess) {
        printf("hipMemset Error: %d %s\n", cudaRes0, hipGetErrorString(cudaRes0));
    }

    //copy input sequences to device
    hipMemcpy(seq1D, a, an * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(seq2D, b, bn * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(seq1_offD, a_offset, subSequenceNum * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(seq2_offD, b_offset, subSequenceNum * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(rowNumsD, ans, subSequenceNum * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(columnNumsD, bns, subSequenceNum * sizeof(int), hipMemcpyHostToDevice);

    cudaRes0 = hipGetLastError();
    if(cudaRes0 != hipSuccess) {
        printf("hipMemcpy Error: %d %s\n", cudaRes0, hipGetErrorString(cudaRes0));
    }
    TIMER_START(0);
    MatchString <<< subSequenceNum, blockSize>>>(
        pathFlagD,
        extFlagD,
        seq1D,
        seq2D,
        seq1_offD,
        seq2_offD,
        openPenalty,
        extensionPenalty,
        match,
        mismatch,
        rowNumsD,
        columnNumsD,
        elementD,
        n_elementD,
        alignment_offsetD,
        maxMatrix
    );
    hipDeviceSynchronize();
    TIMER_END(0);
    swat_kernel_t += MICRO_SECONDS(0);
    cudaRes0 = hipGetLastError();
    if(cudaRes0 != 0)
        printf("Error: %d %s\n", cudaRes0, hipGetErrorString(cudaRes0));

    //copy matrix score structure back
    hipMemcpy(element,  elementD, sizeof(SEQ_ALIGN) * 100 * subSequenceNum, hipMemcpyDeviceToHost);
    hipMemcpy(n_element,  n_elementD, sizeof(int) * subSequenceNum, hipMemcpyDeviceToHost);
    hipMemcpy(alignment_offset,  alignment_offsetD, sizeof(int) * subSequenceNum, hipMemcpyDeviceToHost);

    for(int k = 0; k < subSequenceNum; k++) {
        vector<CigarOp> lce;
        for(int t = n_element[k] - 1; t >= 0; t--) {
            lce.push_back(makeElement(element[t + k * 100].o, element[t + k * 100].length));
        }
        Cigars.push_back(lce);

    }
    hipFree(seq1D);
    hipFree(seq2D);
    hipFree(pathFlagD);
    hipFree(extFlagD);
    hipFree(alignment_offsetD);
    free(element);
    hipFree(elementD);
    free(n_element);
    hipFree(n_elementD);
    return 0;
}


